#include "hip/hip_runtime.h"
#include "caffe/util/add_layer_util.hpp"
#include "caffe/util/device_alternate.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

// kernel methods

template<typename Dtype>
__global__ void add_layers_gpu_kernel(const Dtype *bottom, const Dtype *weight,
                                      Dtype *top, int channels, int num_output,
                                      int height, int width);

template<>
__global__ void add_layers_gpu_kernel<float>(const float *bottom,
                                             const float *weight, float *top,
                                             int channels, int num_output,
                                             int height, int width) {
  const int batch_idx = blockIdx.x;
  const int n = blockIdx.y;
  const int k = blockIdx.z * blockDim.x + threadIdx.x;

  if (k < channels) {
    const float *bottom_data = bottom
        + ((batch_idx * channels + k) * height) * width;
    float *top_data = top + ((batch_idx * num_output + n) * height) * width;

    // Add result on top of existing top (multiply with weight is 1 or 0)
    const float *alpha = weight + (n * channels + k);
    // caffe_gpu_axpy(height * width, alpha, bottom_data, top_data);
  }
}

template<>
__global__ void add_layers_gpu_kernel<double>(const double *bottom,
                                              const double *weight, double *top,
                                              int channels, int num_output,
                                              int height, int width) {

}
// end of kernel methods

template<typename Dtype>
void add_layers_gpu(const Dtype *bottom, const Dtype *weight, Dtype *top,
                    int num, int num_output, int channels, int height,
                    int width) {

  dim3 block_num(num, num_output, channels / CAFFE_CUDA_NUM_THREADS + 1);
  int thread_num = CAFFE_CUDA_NUM_THREADS;

  add_layers_gpu_kernel<<<block_num, thread_num>>>(bottom, weight, top,
                                                   channels, num_output, height,
                                                   width);

}

template void add_layers_gpu<float>(const float *bottom, const float *weight,
                                    float *top, int num, int num_output,
                                    int channels, int height, int width);

template void add_layers_gpu<double>(const double *bottom, const double *weight,
                                     double *top, int num, int num_output,
                                     int channels, int height, int width);

}  // namespace caffe
