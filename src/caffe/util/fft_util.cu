#include "hip/hip_runtime.h"
#include <algorithm>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/fft_util.hpp"
#include <hipfft/hipfft.h>
#include <npp.h>

namespace caffe {
template <typename Dtype>
__global__ void pad_real_blob_gpu_kernel(const int K, const int H, const int W, const int fft_height, const int fft_width,
                                         const int fft_real_size, const Dtype *blob_data, Dtype *padded_data,
                                         const int pad_h, const int pad_w, const bool flip) {

  // blockDim (256, 1) ----- (num_output_, (ch_gr) / CUDA_NUM_THREADS)
  //                              x                y
  const int n = blockIdx.x;

  // calculate the channel index. The blockIdx.y is usually zero. Because CUDA_THREADS = 512 > 48 = ch/gr. (48 / 512) + 1 = 1.
  const int k = blockIdx.y * blockDim.x + threadIdx.x;



  if (k < K) {

    // get offset with channels and the idx of the output.
    const int offset_weight_real = (n * K + k) * fft_real_size;
    const int offset_blob_real = (n * K + k) * H * W;

    for (int h = 0; h < H; h++) {
      for (int w = 0; w < W; w++) {
        // e.g. a 3x3 filter should fit into a 5x5 because the image size is 5x5
        // <--W-->
        // ^ f f f 0 0
        // H f f f 0 0
        // _ f f f 0 0
        //   0 0 0 0 0
        //   0 0 0 0 0

        const int idx_weight_real = offset_weight_real + (h + pad_h) * fft_width + (w + pad_w);
        // copy each weight into the fft_weights_in_real_
        // get ptr to blob data. indexing see: http://caffe.berkeleyvision.org/tutorial/net_layer_blob.html
        // Blob memory is row-major in layout, so the last / rightmost dimension changes fastest. For example,
        // in a 4D blob, the value at index (n, k, h, w) is physically located at index ((n * K + k) * H + h) * W + w.
        // 96 x 3 x 11 x 11 (num_output_, channels_ / group_, kernel_height, width_)

        // if flip = true ==> flip the indices of the weights. Caffe actually does not a convolution but a
        // a cross-correlation according to: https://github.com/BVLC/caffe/issues/2513
        const int h_idx = flip ? H - (h + 1) : h;
        const int w_idx = flip ? W - (w + 1) : w;
        const int idx_weight_in_blob = offset_blob_real + h_idx * W + w_idx;

        Dtype data_in_blob = blob_data[idx_weight_in_blob];
        padded_data[idx_weight_real] = data_in_blob;
      }
    }
  }
}

template __global__ void pad_real_blob_gpu_kernel<float>(const int K, const int H, const int W, const int fft_height, const int fft_width,
                                                         const int fft_real_size, const float *blob_data, float *padded_data,
                                                         const int pad_h, const int pad_w, const bool flip);

template __global__ void pad_real_blob_gpu_kernel<double>(const int K, const int H, const int W, const int fft_height, const int fft_width,
                                                          const int fft_real_size, const double *blob_data, double *padded_data,
                                                          const int pad_h, const int pad_w, const bool flip);

__global__ void fft_pointwise_multiply_float_gpu_kernel(const int N, const int K, const int H, const int W,
                                                        const int weight_group_size, const hipfftComplex *ffted_bottom_data,
                                                        const hipfftComplex *weight_complex, hipfftComplex *ptwise_result) {

  // blockDim (256, 1) ----- (num_output_, (ch_gr) / CUDA_NUM_THREADS)
  //                              x                y
  const int n = blockIdx.x;

  // calculate the channel index. blockIdx.y is of size (H*W/CUDA_NUM_THREADS). So 1 or 2...
  const int hw = blockIdx.y * blockDim.x + threadIdx.x;


  if (hw < H*W) {

    // printf("<<<%i, %i>>>| n: %i k: %i K: %i\n", blockIdx.x, threadIdx.x, n, k, K);
    // check which group_ idx we are in
    const int group_idx = n / weight_group_size;

    // loop over channels
    for (int k = 0; k < K; ++k) {
      // get the input_k. this is the k we use to index the input k-dimension. the max input_k is group_ times more
      // than the max k of the weight.
      const int input_k = k + group_idx * K;
      const int weight_offset = (n * K + k);

      const int input_idx = input_k * H * W + hw;
      const hipfftComplex input = ffted_bottom_data[input_idx];

      const int weight_idx = weight_offset * H * W + hw;
      const hipfftComplex weight = weight_complex[weight_idx];

      // formula for complex mult from here: https://en.wikipedia.org/wiki/Complex_number#Multiplication_and_division
      // (a+bi) (c+di) = (ac-bd) + (bc+ad)i.
      float a = weight.x;
      float b = weight.y;
      float c = input.x;
      float d = input.y;

      const int res_idx = n * H * W + hw;
      ptwise_result[res_idx].x += a * c - b * d;
      ptwise_result[res_idx].y += b * c + a * d;
    }
  }
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__global__ void fft_pointwise_multiply_double_gpu_kernel(const int N, const int K, const int H, const int W,
                                                         const int weight_group_size, const hipfftDoubleComplex *ffted_bottom_data,
                                                         const hipfftDoubleComplex *weight_complex, hipfftDoubleComplex *ptwise_result) {

  // blockDim (256, 1) ----- (num_output_, (ch_gr) / CUDA_NUM_THREADS)
  //                              x                y
  const int n = blockIdx.x;

  // calculate the channel index. The blockIdx.y is usually zero. Because CUDA_THREADS = 512 > 48 = ch/gr. (48 / 512) + 1 = 1.
  const int k = blockIdx.y * blockDim.x + threadIdx.x;


  if (k < K) {
    // check which group_ idx we are in
    const int group_idx = n / weight_group_size;

    // get the input_k. this is the k we use to index the input k-dimension. the max input_k is group_ times more
    // than the max k of the weight.
    const int input_k = k + group_idx * K;
    const int weight_offset = (n * K + k);

    /* in the following loops every filter response is being calculated. there are num_output_ * (channels_ / group_) filters...
     * each (1/group_) part is multiplied with each part of the input. e.g. for group_ = 2, n_o_ 256 and c_ = 96:
     * weights dim: 256x48x5x5, input dim: 1x96x27x27 --> splitted into [1x48x27x27, 1x48x27x27]
     * first 128 weights [128x48x5x5] will be convolved with first part of weights (dimension match!) --> 128 responses
     * same for 2nd part --> 2x128 responses to forward to the next channel
     */
    for (int h = 0; h < H; ++h) {
      for (int w = 0; w < W; ++w) {
        // Indexing: ((n * K + k) * H + h) * W + w
        const int input_idx = (input_k * H + h) * W + w; // 4 ops
        const hipfftDoubleComplex input = ffted_bottom_data[input_idx];

        const int weight_idx = (weight_offset * H + h) * W + w; // 4 ops
        const hipfftDoubleComplex weight = weight_complex[weight_idx];

        // formula for complex mult from here: https://en.wikipedia.org/wiki/Complex_number#Multiplication_and_division
        // (a+bi) (c+di) = (ac-bd) + (bc+ad)i.
        double a = weight.x;
        double b = weight.y;
        double c = input.x;
        double d = input.y;

        const int res_idx = (n * H + h) * W + w; // 4 ops; before with channels: ((n * K + k) * H + h) * W + w;
        atomicAdd(&(ptwise_result[res_idx].x), a * c - b * d);
        atomicAdd(&(ptwise_result[res_idx].y), b * c + a * d);
      }
    }
  }
}

template <typename Dtype>
__global__ void fft_util_normalize_gpu_kernel(const int N, const int H, const int W, const int kernel_h,
                                              const int kernel_w, const int stride_h, const int stride_w,
                                              float normalize_factor, int fft_height, int fft_width,
                                              const Dtype *fft_convolution_result_real, Dtype *top_data) {

  // blockDim (256, 1) ----- (num_output_, (height_out) / CUDA_NUM_THREADS)
  //                              x                y
  const int n = blockIdx.x;

  // calculate the height index. The blockIdx.y is usually zero. Because CUDA_THREADS = 512 > 27 = ch/gr. (27 / 512) + 1 = 1.
  const int h = blockIdx.y * blockDim.x + threadIdx.x;

  if (h < H) {
    const int fft_real_size = fft_height * fft_width;
    const int offset_res_real = n * fft_real_size;
    // caffe does a valid convolution. fft is a full convolution. so the first 'valid' result is at
    // idx (kernel_h_ - 1). The stride times the idx of the output pixel will be added onto this.
    const int h_idx = (kernel_h - 1) + h * stride_h;

    for (int w = 0; w < W; ++w) // =55 in 1st layer
    {
      // caffe does a valid convolution. fft is a full convolution. so the first 'valid' result is at
      // idx (kernel_w_ - 1). The stride times the idx of the output pixel will be added onto this.
      const int w_idx = (kernel_w - 1) + w * stride_w;
      //((n * K + k) * H + h) * W + w;
      const int top_data_idx = (n * H + h) * W + w;

      // the index in the data of the convolution result array (the real one)
      const int res_data_idx = offset_res_real + h_idx * fft_width + w_idx;

      // normalize fft and sum up everything from the input channels...
      top_data[top_data_idx] = fft_convolution_result_real[res_data_idx] * normalize_factor;
    }
  }
}

template __global__ void fft_util_normalize_gpu_kernel<float>(const int N, const int H, const int W, const int kernel_h,
                                                              const int kernel_w, const int stride_h, const int stride_w,
                                                              float normalize_factor, int fft_height, int fft_width,
                                                              const float *fft_convolution_result_real, float *top_data);

template __global__ void fft_util_normalize_gpu_kernel<double>(const int N, const int H, const int W, const int kernel_h,
                                                               const int kernel_w, const int stride_h, const int stride_w,
                                                               float normalize_factor, int fft_height, int fft_width,
                                                               const double *fft_convolution_result_real, double *top_data);


//// --- end of kernel methods ---

template <>
void npp_complex_add_product<float>(const std::complex<float> *src1, const std::complex<float> *src2, std::complex<float> *dst, int len)
{
  NPP_CHECK(nppsAddProduct_32fc(reinterpret_cast<const Npp32fc *> (src1),
                                reinterpret_cast<const Npp32fc *> (src2),
                                reinterpret_cast<Npp32fc *> (dst), len));
}

template <>
void npp_complex_add_product<double>(const std::complex<double> *src1, const std::complex<double> *src2, std::complex<double> *dst, int len)
{

  NPP_CHECK(nppsAddProduct_64fc(reinterpret_cast<const Npp64fc *> (src1),
                                reinterpret_cast<const Npp64fc *> (src2),
                                reinterpret_cast<Npp64fc *> (dst), len));
}


template <typename Dtype>
void pad_real_blob_gpu(std::vector<int> shape, const int fft_height, const int fft_width,
                       const Dtype *blob_data, Dtype *padded_data, const int pad_h,
                       const int pad_w, const bool flip) {

  const int N = shape[0];
  const int K = shape[1];
  const int H = shape[2];
  const int W = shape[3];

  const int fft_real_size = fft_height * fft_width;

  int num_arr = N * K; // # of arrays (for weights it is num_weights [96 x 3]
  // for input data it is channels [ 1 x 3]

  // set everything to 0 before --> so not set weights are 0-padded :)
  caffe_gpu_memset(fft_real_size * num_arr * sizeof(Dtype), 0., padded_data);

  // N = 256 (num_output_)
  // K = 96 / 2 (channels / group) ==> (48 / 512 ) + 1 = 1
  dim3 block_num(N, (K / CAFFE_CUDA_NUM_THREADS) + 1);
  int thread_num = CAFFE_CUDA_NUM_THREADS;

  pad_real_blob_gpu_kernel<Dtype><<<block_num, thread_num>>>(
      K, H, W, fft_height, fft_width, fft_real_size,
      blob_data, padded_data, pad_h, pad_w, flip);
  CUDA_POST_KERNEL_CHECK;
}

template void pad_real_blob_gpu<float>(std::vector<int> shape, const int fft_height, const int fft_width,
                                       const float *blob_data, float *padded_data, const int pad_h,
                                       const int pad_w, const bool flip);

template void pad_real_blob_gpu<double>(std::vector<int> shape, const int fft_height, const int fft_width,
                                        const double *blob_data, double *padded_data, const int pad_h,
                                        const int pad_w, const bool flip);

template <>
void fft_util_pointwise_multiply_gpu<float>(std::vector<int> shape, int group, const std::complex<float> *ffted_bottom_data,
                                            const std::complex<float> *weight_complex, std::complex<float> *ptwise_result) {
  const int N = shape[0];
  const int K = shape[1];
  const int H = shape[2];
  const int W = shape[3];

  const int weight_group_size = N / group;

  // N = 256 (num_output_)
  // K = 96 / 2 (channels / group) ==> (48 / 512 ) + 1 = 1
  // dim3 block_num(N, (K / CAFFE_CUDA_NUM_THREADS) + 1);

  // N = num_output, H * W as second dim so no races happen; because over K (channels) will be summed up
  // On thoise channels sum -ups the threads interfere with another...
  dim3 block_num(N, (H * W / CAFFE_CUDA_NUM_THREADS) + 1);
  int thread_num = CAFFE_CUDA_NUM_THREADS;

  const hipfftComplex *ffted_bottom_data_cuda  = reinterpret_cast<const hipfftComplex *> (ffted_bottom_data);
  const hipfftComplex *weight_complex_cuda = reinterpret_cast<const hipfftComplex *> (weight_complex);
  hipfftComplex *ptwise_result_cuda = reinterpret_cast<hipfftComplex *> (ptwise_result);

  fft_pointwise_multiply_float_gpu_kernel<<<block_num, thread_num>>>
      (N, K, H, W, weight_group_size, ffted_bottom_data_cuda, weight_complex_cuda, ptwise_result_cuda);
  CUDA_POST_KERNEL_CHECK;
}

template <>
void fft_util_pointwise_multiply_gpu<double>(std::vector<int> shape, int group, const std::complex<double> *ffted_bottom_data,
                                             const std::complex<double> *weight_complex, std::complex<double> *ptwise_result) {
  const int N = shape[0];
  const int K = shape[1];
  const int H = shape[2];
  const int W = shape[3];

  const int weight_group_size = N / group;

  // N = 256 (num_output_)
  // K = 96 / 2 (channels / group) ==> (48 / 512 ) + 1 = 1
  dim3 block_num(N, (K / CAFFE_CUDA_NUM_THREADS) + 1);
  int thread_num = CAFFE_CUDA_NUM_THREADS;

  const hipfftDoubleComplex *ffted_bottom_data_cuda  = reinterpret_cast<const hipfftDoubleComplex *> (ffted_bottom_data);
  const hipfftDoubleComplex *weight_complex_cuda = reinterpret_cast<const hipfftDoubleComplex *> (weight_complex);
  hipfftDoubleComplex *ptwise_result_cuda = reinterpret_cast<hipfftDoubleComplex *> (ptwise_result);

  fft_pointwise_multiply_double_gpu_kernel<<<block_num, thread_num>>>
      (N, K, H, W, weight_group_size, ffted_bottom_data_cuda, weight_complex_cuda, ptwise_result_cuda);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void fft_util_pointwise_multiply_npp_gpu(std::vector<int> shape, int group, const std::complex<Dtype> *bottom_complex,
                                         const std::complex<Dtype> *weight_complex, std::complex<Dtype> *ptwise_result) {
  const int N = shape[0];
  const int K = shape[1];
  const int H = shape[2];
  const int W = shape[3];

  const int weight_group_size = N / group;
  const int fft_complex_size = H * W;

  int n = 0;
  int k = 0;

  for (n = 0; n < N; ++n) {
    const int res_offset = n * fft_complex_size;
    // check which group_ idx we are in
    const int group_idx = n / weight_group_size;
    for (k = 0; k < K; ++k) {
      // get the input_k. this is the k we use to index the input k-dimension. the max input_k is group_ times more
      // than the max k of the weight.
      const int input_offset = (k + group_idx * K) * fft_complex_size;
      const int weight_offset = (n * K + k) * fft_complex_size;
      npp_complex_add_product<Dtype>(bottom_complex + input_offset, weight_complex + weight_offset,
                                     ptwise_result + res_offset, fft_complex_size);
    }
  }
}

template void fft_util_pointwise_multiply_npp_gpu<float>(std::vector<int> shape, int group,
                                                         const std::complex<float> *ffted_bottom_data,
                                                         const std::complex<float> *weight_complex,
                                                         std::complex<float> *ptwise_result);

template void fft_util_pointwise_multiply_npp_gpu<double>(std::vector<int> shape, int group,
                                                          const std::complex<double> *ffted_bottom_data,
                                                          const std::complex<double> *weight_complex,
                                                          std::complex<double> *ptwise_result);

template <typename Dtype>
void fft_util_normalize_gpu(std::vector<int> shape, const int kernel_h,
                            const int kernel_w, const int stride_h, const int stride_w,
                            float normalize_factor, int fft_height, int fft_width,
                            const Dtype *conv_result_real, Dtype *top_data) {

  // shape[0] is 0 here, because there is only one output image.
  const int N = shape[1];
  const int H = shape[2];
  const int W = shape[3];

  dim3 block_num(N, (H / CAFFE_CUDA_NUM_THREADS) + 1);
  int thread_num = CAFFE_CUDA_NUM_THREADS;


  fft_util_normalize_gpu_kernel<<<block_num, thread_num>>>
      (N, H, W, kernel_h, kernel_w, stride_h, stride_w,
       normalize_factor, fft_height, fft_width,
       conv_result_real, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template void fft_util_normalize_gpu<float>(std::vector<int> shape, const int kernel_h,
                                            const int kernel_w, const int stride_h, const int stride_w,
                                            float normalize_factor, int fft_height, int fft_width,
                                            const float *conv_result_real, float *top_data);

template void fft_util_normalize_gpu<double>(std::vector<int> shape, const int kernel_h,
                                            const int kernel_w, const int stride_h, const int stride_w,
                                            float normalize_factor, int fft_height, int fft_width,
                                            const double *conv_result_real, double *top_data);

// --- cufft calls ----


template<>
void fft_gpu_plan_many_dft_r2c_2d<float>(hipfftHandle *plan, int n0,
                                         int n1,
                                         int how_many) {
  int rank = 2;
  int n[] = {n0, n1};
  int idist = n0 * n1; /* = 256*256, the distance in memory
                                          between the first element
                                          of the first array and the
                                          first element of the second array */
  int istride = 1; /* array is contiguous in memory */
  int *inembed = NULL;

  // out
  int odist = n0 * (n1 / 2 + 1);
  int ostride = 1;
  int *onembed = NULL;

  CUFFT_CHECK(hipfftCreate(plan));
  CUFFT_CHECK(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, how_many));
}

template<>
void fft_gpu_plan_many_dft_r2c_2d<double>(hipfftHandle *plan, int n0,
                                          int n1,
                                          int how_many) {
  int rank = 2;
  int n[] = {n0, n1};
  int idist = n0 * n1; /* = 256*256, the distance in memory
                                          between the first element
                                          of the first array and the
                                          first element of the second array */
  int istride = 1; /* array is contiguous in memory */
  int *inembed = NULL;

  // out
  int odist = n0 * (n1 / 2 + 1);
  int ostride = 1;
  int *onembed = NULL;

  CUFFT_CHECK(hipfftCreate(plan));
  CUFFT_CHECK(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_D2Z, how_many));
}

template<>
void fft_gpu_plan_many_dft_c2r_2d<float>(hipfftHandle *plan, int n0,
                                         int n1,
                                         int how_many) {

  int rank = 2;
  int n[] = {n0, n1};
  int idist = n0 * (n1 / 2 + 1); /* = 256*129, the distance in memory
                                          between the first element
                                          of the first array and the
                                          first element of the second array */
  int istride = 1; /* array is contiguous in memory */
  int *inembed = NULL;

  // out
  int odist = n0 * n1;
  int ostride = 1;
  int *onembed = NULL;

  CUFFT_CHECK(hipfftCreate(plan));
  CUFFT_CHECK(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2R, how_many));
}

template<>
void fft_gpu_plan_many_dft_c2r_2d<double>(hipfftHandle *plan, int n0,
                                         int n1,
                                         int how_many) {

  int rank = 2;
  int n[] = {n0, n1};
  int idist = n0 * (n1 / 2 + 1); /* = 256*129, the distance in memory
                                          between the first element
                                          of the first array and the
                                          first element of the second array */
  int istride = 1; /* array is contiguous in memory */
  int *inembed = NULL;

  // out
  int odist = n0 * n1;
  int ostride = 1;
  int *onembed = NULL;

  CUFFT_CHECK(hipfftCreate(plan));
  CUFFT_CHECK(hipfftPlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, how_many));
}

template<>
void fft_gpu_execute_plan_r2c<float>(hipfftHandle plan, float *in, std::complex<float> *out) {
  CUFFT_CHECK(hipfftExecR2C(plan, in, reinterpret_cast<hipfftComplex *>(out)));
}

template<>
void fft_gpu_execute_plan_r2c<double>(hipfftHandle plan, double *in, std::complex<double> *out) {
  CUFFT_CHECK(hipfftExecD2Z(plan, in, reinterpret_cast<hipfftDoubleComplex *>(out)));
}

template<>
void fft_gpu_execute_plan_c2r<float>(hipfftHandle plan, std::complex<float> *in, float *out) {
  CUFFT_CHECK(hipfftExecC2R(plan, reinterpret_cast<hipfftComplex *>(in), out));
}

template<>
void fft_gpu_execute_plan_c2r<double>(hipfftHandle plan, std::complex<double> *in, double *out) {
  CUFFT_CHECK(hipfftExecZ2D(plan, reinterpret_cast<hipfftDoubleComplex *>(in), out));
}

void fft_gpu_destroy_plan(hipfftHandle plan_handle) {
  hipfftDestroy(plan_handle);
}

}
