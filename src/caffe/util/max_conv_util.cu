#include "hip/hip_runtime.h"
#include "caffe/util/max_conv_util.hpp"
#include "caffe/util/device_alternate.hpp"
#include <float.h>
#include <stdio.h>

namespace caffe {

// kernel methods

template<typename Dtype>
__device__ Dtype max(Dtype a, Dtype b);

template<>
__device__ float max<float>(float a, float b) {
  return fmaxf(a, b);
}

template<>
__device__ double max<double>(double a, double b) {
  return fmax(a, b);
}

template<typename Dtype>
__global__ void max_convolution_gpu_kernel(const Dtype *bottom,
                                           const Dtype *weight, Dtype *top,
                                           Dtype *top_origin,
                                           int kernel_h, int kernel_w,
                                           int kernel_radius, int channels,
                                           int height, int width) {
  const int batch_idx = blockIdx.x;
  const int k = blockIdx.y;
  const int hw = blockIdx.z * blockDim.x + threadIdx.x;

  if (hw < height * width) {
    // do sth...
    int y = hw / width;
    int x = hw % width;

    const Dtype *bottom_data = bottom
        + ((batch_idx * channels + k) * height) * width;
    const Dtype *kernel_weight_data = weight + k * kernel_h * kernel_w;
    Dtype *top_data = top + ((batch_idx * channels + k) * height) * width;

    Dtype max_val = FLT_MIN;
    int max_y = y;
    int max_x = x;
    for (int q = 0; q < kernel_h; ++q) {
      for (int r = 0; r < kernel_w; ++r) {
        int y_bottom = y + (q - kernel_radius);
        int x_bottom = x + (r - kernel_radius);

        // ignore borders...
        if (!(y_bottom < 0 || x_bottom < 0 || y_bottom >= height
            || x_bottom >= width)) {
          // - for max conv and + for min conv ???
          Dtype tmp = bottom_data[y_bottom * width + x_bottom]
              - kernel_weight_data[q * kernel_w + r];
          
          if (tmp > max_val) {
            // here we also remember where the maximum came from
            max_val = tmp;
            max_y = y_bottom;
            max_x = x_bottom;
          }
        }
      }
    }
    top_data[y * width + x] = max_val;
    
    // * 2 because of the 5-th dim in the blob.
    Dtype *top_coord = top_origin + (((batch_idx * channels + k) * height + y) * width + x) * 2;
    top_coord[0] = max_y;
    top_coord[1] = max_x;
  }
}

template
__global__ void max_convolution_gpu_kernel<float>(const float *bottom,
                                                  const float *weight,
                                                  float *top,
                                                  float *top_origin,
                                                  int kernel_h,
                                                  int kernel_w, int kernel_radius,
                                                  int channels, int height,
                                                  int width);

template
__global__ void max_convolution_gpu_kernel<double>(const double *bottom,
                                                   const double *weight,
                                                   double *top,
                                                   double *top_origin,
                                                   int kernel_h,
                                                   int kernel_w,
                                                   int kernel_radius,
                                                   int channels, int height,
                                                   int width);

template<typename Dtype>
__global__ void fast_max_convolution_gpu_kernel(const Dtype *bottom,
                                                const Dtype *weight, Dtype *top,
                                                Dtype *top_origin,
                                                int kernel_h, int kernel_w,
                                                int kernel_radius, int channels,
                                                int height, int width,
                                                int tile_dim, int tiles_per_row, int shared_mem_length) {
  const int batch_idx = blockIdx.x;
  const int k = blockIdx.y;

  const Dtype *bottom_data = bottom
      + ((batch_idx * channels + k) * height) * width;

  // tiles are quadratically... get initial offset...
  const int tile_idx = blockIdx.z; // * blockDim.x + threadIdx.x;
  const int y_tile_idx = (tile_idx / tiles_per_row);
  const int x_tile_idx = (tile_idx % tiles_per_row);

  const int y_offset = y_tile_idx * tile_dim;
  const int x_offset = x_tile_idx * tile_dim;

  const int y_start = y_offset - kernel_radius;
  const int x_start = x_offset - kernel_radius;

  // we have to copy the values from offset (X) into shared mem:
  // *************** (kernel_radius) **************|stride! = width|
  // (kernel_radius)X --- tile_dim -▶(kernel_radius)
  // ***************|                **************
  // ***************| tile_dim       **************
  // ***************▼               ◢**************
  // *************** (kernel_radius) **************


  // We store a tile of the bottom data here + the kernel
  extern __shared__ float shared_mem[];
  Dtype *shared_data = (Dtype *)&shared_mem;

  // every thread has to copy some amount of values from global to shared memory
  const int shared_data_size = shared_mem_length * shared_mem_length;
  const int copy_count = (shared_data_size / blockDim.x) + 1; // calculated outside !? e.g. [(50*50)/512] + 1 = 5

  // offset of current thread inside the shared memory...
  const int inner_offset = threadIdx.x * copy_count;

  for (int i = 0; i < copy_count; ++i) {
    const int copy_offset = (inner_offset + i);
    const int shared_mem_y_offset = copy_offset / shared_mem_length;
    const int shared_mem_x_offset = copy_offset % shared_mem_length;

    // case III: we are at the end of a tile (including apron!) --> set x to x_start and current y to +=1
    const int src_y = y_start + shared_mem_y_offset; // handles case III
    const int src_x = x_start + shared_mem_x_offset; // handles case III

   // don't copy data outside of the allocated space
    if (copy_offset < shared_data_size) {
      // find destination ptr
      Dtype *dst = shared_data + copy_offset;
      // case I  : we are at the end of a row/column in the image? --> fill 0s into shared mem
      // case II : we are before the beginning of a row/column in a the image? --> fill 0s into shared mem
      if (src_y < 0 || src_x < 0 || src_x >= width || src_y >= height) {
        *dst = 0.;
      } else {
        *dst = bottom_data[(src_y * width) + src_x];
      }
    }
  }

  // write kernel to shared mem
  // ------------------------------
  // we use one array in the shared memory for both data and kernel (we only dynamically alloc one array from outside the kernel)
  Dtype *kernel = shared_data + shared_data_size;
  const int k_shared_mem_size = kernel_h * kernel_w;
  const int k_copy_count = (k_shared_mem_size / blockDim.x) + 1; // calculated outside !? e.g. [(50*50)/512] + 1 = 5
  // offset of current thread inside the kernel shared memory...
  const int k_inner_offset = threadIdx.x * k_copy_count;

  // offset to kernel
  const Dtype *kernel_weight_data = weight + k * kernel_h * kernel_w;

  for (int i = 0; i < k_copy_count; ++i) {
    int idx = k_inner_offset + i;
    if (idx < k_shared_mem_size) {
      kernel[idx] = kernel_weight_data[idx];
    }
  }

  // sync shared memory...
  __syncthreads();

  const int hw = threadIdx.x;

  if (hw < tile_dim * tile_dim) {
    int y = (hw / tile_dim);
    int x = (hw % tile_dim);

    // we write in global memory, because we only access each location once
    int top_y = y + y_offset;
    int top_x = x + x_offset;

    Dtype max_val = FLT_MIN;
    int max_y = top_y;
    int max_x = top_x;
    for (int q = 0; q < kernel_h; ++q) {
      for (int r = 0; r < kernel_w; ++r) {
        int y_bottom = y + q;
        int x_bottom = x + r;

          // - for max conv and + for min conv ???
          Dtype tmp = shared_data[y_bottom * shared_mem_length + x_bottom]
              - kernel[q * kernel_w + r];

          if (tmp > max_val) {
            // here we also remember where the maximum came from
            max_val = tmp;
            max_y = y_bottom + y_start;
            max_x = x_bottom + x_start;
          }
      }
    }

    // check if access is outside of valid bounds.
    if (top_y < height && top_x < width) {
      Dtype *top_data = top + ((batch_idx * channels + k) * height) * width;
      const int top_idx = top_y * width + top_x;
      top_data[top_idx] = max_val;

      // * 2 because of the 5-th dim in the blob.
      Dtype *top_coord = top_origin + (((batch_idx * channels + k) * height + top_y) * width + top_x) * 2;
      top_coord[0] = max_y;
      top_coord[1] = max_x;
    }
  }
}


template
__global__ void fast_max_convolution_gpu_kernel<float>(const float *bottom,
                                                       const float *weight, float *top,
                                                       float *top_origin,
                                                       int kernel_h, int kernel_w,
                                                       int kernel_radius, int channels,
                                                       int height, int width,
                                                       int tile_dim, int tiles_per_row, int shared_mem_length);


template
__global__ void fast_max_convolution_gpu_kernel<double>(const double *bottom,
                                                        const double *weight, double *top,
                                                        double *top_origin,
                                                        int kernel_h, int kernel_w,
                                                        int kernel_radius, int channels,
                                                        int height, int width,
                                                        int tile_dim, int tiles_per_row, int shared_mem_length);


// end of kernel methods

template<typename Dtype>
void max_convolution_gpu(const Dtype *bottom, const Dtype *weight, Dtype *top, Dtype *top_origin,
                         int kernel_h, int kernel_w, int num, int channels,
                         int height, int width) {
  dim3 block_num(num, channels, (height * width) / CAFFE_CUDA_NUM_THREADS);
  int thread_num = CAFFE_CUDA_NUM_THREADS; //CAFFE_CUDA_NUM_THREADS;

  int k_radius = (kernel_h - 1) / 2;  // we only support quadratic kernels and assume uneven filter sizes...

  max_convolution_gpu_kernel<<<block_num, thread_num>>>(bottom, weight, top, top_origin,
                                                        kernel_h, kernel_w,
                                                        k_radius, channels,
                                                        height, width);

}

template void max_convolution_gpu<float>(const float *bottom,
                                         const float *weight, float *top, float *top_origin,
                                         int kernel_h, int kernel_w, int num,
                                         int channels, int height, int width);

template void max_convolution_gpu<double>(const double *bottom,
                                          const double *weight, double *top, double *top_origin,
                                          int kernel_h, int kernel_w, int num,
                                          int channels, int height, int width);

template<typename Dtype>
void fast_max_convolution_gpu(const Dtype *bottom, const Dtype *weight, Dtype *top, Dtype *top_origin,
                         int kernel_h, int kernel_w, int num, int channels,
                         int height, int width) {

#define TITAN_NUM_THREADS 1024

  int tile_dim = sqrt(TITAN_NUM_THREADS);
  const int tiles_per_row = width / tile_dim + 1;
  const int tiles_per_col = height / tile_dim + 1;
  int z_blocks = (tiles_per_col  * tiles_per_row);

  dim3 block_num(num, channels, z_blocks);
  int thread_num = TITAN_NUM_THREADS;

  int k_radius = (kernel_h - 1) / 2;  // we only support quadratic kernels and assume uneven filter sizes...

  const int shared_mem_length = (tile_dim + 2 * k_radius);
  // size of bottom data region + kernel region
  int shared_mem_size = (shared_mem_length * shared_mem_length + kernel_h * kernel_w) * sizeof(Dtype);

  fast_max_convolution_gpu_kernel<<<block_num, thread_num, shared_mem_size>>>(bottom, weight, top, top_origin,
                                                            kernel_h, kernel_w,
                                                            k_radius, channels,
                                                            height, width, tile_dim, tiles_per_row, shared_mem_length);

}

template void fast_max_convolution_gpu<float>(const float *bottom,
                                              const float *weight, float *top, float *top_origin,
                                              int kernel_h, int kernel_w, int num,
                                              int channels, int height, int width);

template void fast_max_convolution_gpu<double>(const double *bottom,
                                               const double *weight, double *top, double *top_origin,
                                               int kernel_h, int kernel_w, int num,
                                               int channels, int height, int width);

}  // namespace caffe
